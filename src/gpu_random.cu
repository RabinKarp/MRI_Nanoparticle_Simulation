
/*
 * This program uses the host CURAND API to generate 100
 * pseudorandom floats.
 */

#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "gpu_random.h"

using namespace std;

gpu_rng::gpu_rng() {
    hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_XORWOW);
    
    /* Set the seed as the current time*/
    hiprandSetPseudoRandomGeneratorSeed(gen, time(NULL));

}

gpu_rng::~gpu_rng() {
    hiprandDestroyGenerator(gen);
}

void gpu_rng::getUniformDoubles(size_t n, double *devData)
{
    hiprandGenerateUniformDouble(gen, devData, n);
}


void gpu_rng::getNormalDoubles(size_t n, double *devData)
{
    hiprandGenerateNormalDouble(gen, devData, n, 0.0, 1.0);
}
