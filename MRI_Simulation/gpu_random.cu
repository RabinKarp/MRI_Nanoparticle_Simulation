
/*
 * This program uses the host CURAND API to generate 100
 * pseudorandom floats.
 */
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "cuda_helpers.h"

using namespace std;

void getUniformDoubles(size_t n, double *devData)
{
    hiprandGenerator_t gen;;

    hipEvent_t start, stop;

    /* Create pseudo-random number generator */
    /*CURAND_CALL(hiprandCreateGenerator(&gen,
                HIPRAND_RNG_PSEUDO_DEFAULT));*/
    hiprandCreateGenerator(&gen,
                HIPRAND_RNG_PSEUDO_XORWOW);

    /* Set the seed as the current time*/
    hiprandSetPseudoRandomGeneratorSeed(gen,
                time(NULL));

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    HANDLE_ERROR(hipEventRecord(start, 0));

    cout << "Generating uniform random numbers!" << endl;

    /* Generate n doubles on device */
    hiprandGenerateUniformDouble(gen, devData, n);
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));

    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    cout << "RNG complete! Elapsed time: "
      << elapsedTime << " ms" << endl;

    /* Cleanup */
    hiprandDestroyGenerator(gen);
}


void getNormalDoubles(size_t n, double *devData)
{
    hiprandGenerator_t gen;;

    hipEvent_t start, stop;

    /* Create pseudo-random number generator */
    /*CURAND_CALL(hiprandCreateGenerator(&gen,
                HIPRAND_RNG_PSEUDO_DEFAULT));*/
    hiprandCreateGenerator(&gen,
                HIPRAND_RNG_PSEUDO_XORWOW);

    /* Set the seed as the current time*/
    hiprandSetPseudoRandomGeneratorSeed(gen,
                time(NULL));

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    HANDLE_ERROR(hipEventRecord(start, 0));

    cout << "Generating normal random numbers!" << endl;

    /* Generate n doubles on device */
    hiprandGenerateNormalDouble(gen, devData, n, 0.0, 1.0);
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));

    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    cout << "RNG complete! Elapsed time: "
      << elapsedTime << " ms" << endl;

    /* Cleanup */
    hiprandDestroyGenerator(gen);
}
