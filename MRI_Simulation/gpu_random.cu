
/*
 * This program uses the host CURAND API to generate 100
 * pseudorandom floats.
 */
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "cuda_helpers.h"

using namespace std;

void getUniformDoubles(size_t n, double *devData)
{
    hiprandGenerator_t gen;;

    /* Create pseudo-random number generator */
    /*CURAND_CALL(hiprandCreateGenerator(&gen,
                HIPRAND_RNG_PSEUDO_DEFAULT));*/
    hiprandCreateGenerator(&gen,
                HIPRAND_RNG_PSEUDO_XORWOW);

    /* Set the seed as the current time*/
    hiprandSetPseudoRandomGeneratorSeed(gen,
                time(NULL));

    /* Generate n doubles on device */
    hiprandGenerateUniformDouble(gen, devData, n);

    /* Cleanup */
    hiprandDestroyGenerator(gen);
}


void getNormalDoubles(size_t n, double *devData)
{
    hiprandGenerator_t gen;;

    /* Create pseudo-random number generator */
    /*CURAND_CALL(hiprandCreateGenerator(&gen,
                HIPRAND_RNG_PSEUDO_DEFAULT));*/
    hiprandCreateGenerator(&gen,
                HIPRAND_RNG_PSEUDO_XORWOW);

    /* Set the seed as the current time*/
    hiprandSetPseudoRandomGeneratorSeed(gen,
                time(NULL));

    /* Generate n doubles on device */
    hiprandGenerateNormalDouble(gen, devData, n, 0.0, 1.0);

    /* Cleanup */
    hiprandDestroyGenerator(gen);
}
