#include "hip/hip_runtime.h"

#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>

#include <iostream>
#include <fstream>
#include <cstdlib>
#include <cmath>
#include <cunistd>
#include <cstdio>

#include "rand_walk.h"
#include "cuda_helpers.h"

using namespace std;

const int num_blocks = 32;
const int threads_per_block = 512;

double in_stdev = sqrt(pi * D_in * tau);
double out_stdev = sqrt(pi * D_out * tau);
double reflectIO = 1 - sqrt(tau / (6*D_in)) * 4 * P_expr;
double reflectOI = 1 - ((1 - reflectIO) * sqrt(D_in/D_out));

const double g = 42.5781e6;             // gyromagnetic ratio in MHz/T

const int spheres_per_cube = 14;
// NOTE: This needs to be copied over to the GPU, can be stored in constant
// memory
const int offsets[spheres_per_cube] = {
    {0, 0, 0}, {2, 0, 0}, {0, 2, 0},
    {2, 2, 0}, {1, 1, 0}, {1, 0, 1},
    {0, 1, 1}, {2, 1, 1}, {1, 2, 1},
    {0, 0, 2}, {2, 0, 2}, {0, 2, 2},
    {2, 2, 2}, {1, 1, 2}};

#define MAX 500000

/**
 * Struct containing all pointers passed from the CPU (host) to the GPU (device)
 * and other information that the GPU needs, e.g. an array of states for each
 * thread.
 */
struct GPUData {
    int num_mnps;
    MNP_info* dev_mnp;

    double* dev_lattice;
    int* sphereLookup;
    water_info* dev_waters;

    unsigned int seed;
    hiprandState_t* states;

    /**
     * The array of magnetizations is a double array of dimension
     * (t * num_blocks). Each block writes to a unique portion of the shared
     * global memory.
     */
     double* magnetizations;
};

/**
 * Prepares data in proper format, loads data onto the GPU and allocates
 * space for the answers that the GPU prepares to compute.
 */
GPUData initializeGPU(vector<MNP_info> *mnpList, water_info* w) {
    GPUData d;
    d.num_mnps = mnpList->size();

    // TODO: Seed the random number generator with the system time!

    int totalThreads = num_blocks * threads_per_block;
    // Initialize a set of random states on the devce
    hipMalloc((void**) &(d.states), totalThreads * sizeof(hiprandState_t));

    // Get data into standard C arrays on the host
    MNP_info* mnps = calloc(mnpList->size() , sizeof(MNP_info));
    double* latticePoints = calloc(num_cells * 3, sizeof(double));

    for(int i = 0; i < num_cells * 3; i++) {
        latticePoints[i * 3] = fcc[i][0];
        latticePoints[i * 3 + 1] = fcc[i][1];
        latticePoints[i * 3 + 2] = fcc[i][2];
    }

    for(int i = 0; i < mnpList->size(); i++) {
        mnps[i] = mnpList[i];
    }

    // Allocate the input data on the GPU
    HANDLE_ERROR(hipMalloc((void **) &(d.dev_mnp),
        sizeof(MNP_info) * d.num_mnps));
    HANDLE_ERROR(hipMalloc((void **) &(d.dev_waters),
        sizeof(water_info) * num_water));
    HANDLE_ERROR(hipMalloc((void **) &(d.dev_lattice),
        sizeof(MNP_info) * d.num_mnps));

    // Allocate the output data on the GPU
    HANDLE_ERROR(hipMalloc((void **) &(d.magnetizations),
        sizeof(double) * t * num_blocks));

    // Copy waters, MNPs, and cells to device
    HANDLE_ERROR(hipMemcpy((void *) mnps, (void *) d.dev_mnp, sizeof(MNP_info) * d.num_mnps,
        hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy((void*) w, (void *) d.dev_waters, sizeof(water_info) * num_water,
        hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy((void *) latticePoints, (void *) d.dev_lattice,
        sizeof(double) * num_cells * 3,
        hipMemcpyHostToDevice));

    // Free memory allocated within this function
    free(latticePoints);
    free(mnps);

    // Return the struct containing device pinters
    return data;
}

/**
 * Computes the array of net magnetizations produced by the GPU and frees
 * all resources allocated on the device.
 */
double* finalizeGPU(GPUData d) {
    double* computedMagnetizations = calloc(t * num_blocks, sizeof(double));
    double* netMagnetizations = calloc(t, sizeof(double));

    // Copy the device memory to the host
    HANDLE_ERROR(hipMemcpy((void *) computedMagnetizations, (void *) d.magnetizations,
        sizeof(double) * t * num_blocks,
        hipMemcpyDeviceToHost));

    // Compute the array of net magnetizations by summing up the data
    // from all blocks for each time step
    for(int i = 0 ; i < t; i++) {
        for(int j = 0; j < num_blocks; j++) {
            netMagnetizations[i] += computedMagnetizations[i * num_blocks + j];
        }
    }

    // Free device memory
    // TODO: Need to complete this section and clean up ALL device memory!
    hipFree(d.dev_mnp);
    hipFree(d.dev_lattice);
    hipFree(d.dev_waters);
    hipFree(d.magnetizations);

    // Free host memory allocated in this function
    free(computedMagnetizations);
    return netMagnetizations
}

/**
 * Initializes the random state associated with each thread - used so that
 * each thread can generate its own random numbers.
 */
__device__ void initRandomState(int tid, unsigned int seed, hiprandState_t* states) {
    hiprand_init(seed, tid, 0, states + tid);
}

/**
 * Returns a double randomly and uniformly distributed from 0 to 1.
 */
__device__ double getUniformDouble(hiprandState_t* state) {
    return hiprand_uniform_double(state);
}

/**
 * Returns a double from a standard normal distribution with the given
 * standard deviation.
 *
 * TODO: Check whether this is the correct way to scale a normal distribution
 */
__device__ void getNormalDouble(hiprandState_t* state, double stdev) {
    return hiprand_normal_double(state) * stdev;
}

__device__ bool in_cell(water_info *w) {
    double *center = fcc[w->nearest];
    double x = w.x - center[0];
    double y = w.y - center[1];
    double z = w.z - center[2];
    return cell_r * cell_r > NORMSQ(x, y, z);
}

/**
 * Updates the cell closest to a water molecule, which is stored by that
 * molecule for easy reference. We use a sphere lookup hash to do so.
 */
__device__ void updateNearestCell(water_info *w, dev_lattice* lattice) {
    // Scale and integerize the coordinates
    double x = ((int) (w->x / (cell_r * 4) * sqrt(2)))*2;
    double y = ((int) (w->y / (cell_r * 4) * sqrt(2)))*2;
    double z = ((int) (w->z / (cell_r * 4) * sqrt(2)))*2;

    double cDist = MAX;
    for(int i = 0; i < spheres_per_cube; i++) {
        int idx = sphereLookup[x + offsets[i][1]][y + offsets[i][2]][[z + offsets[i][3]]];
        double dx = lattice[idx][0] - w->x;
        double dy = lattice[idx][1] - w->y;
        double dz = lattice[idx][2] - w->z;
        if(NORMSQ(dx, dy, dz) < cDist) {
            w->nearest = idx;
            cDist = NORMSQ(dx, dy, dz);
        }
    }
}

/**
 * Returns the random displacement of a water molecule according to a specified
 * normal distribution.
 */
__device__ water_info rand_displacement(water_info *w, hiprandState_t* states) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;

    water_info disp;
    double norm;
    if(in_cell(w)) {
        norm = getNormalDouble(states + tid, in_stdev);
    }
    else {
        norm = getNormalDouble(states + tid, out_stdev);
    }
    disp.x = getUniformDouble(states + tid);
    disp.y = getUniformDouble(states + tid);
    dips.z = getUniformDouble(states + tid);

    double nConstant = NORMSQ(disp.x, disp.y, disp.z);

    disp.x *= norm / nConstant;
    disp.y *= norm / nConstant;
    disp.z *= norm / nConstant;

    return disp;
}

/**
 * Check for cell boundary crossings and apply a reflection appropriately
 *
 * Returns true if a reflection occurs and false otherwise.
 *
 */
__device__ bool cellReflection(water_info* i, water_info* f) {

    bool ret =
    // First handle the case where the water diffuses into the cell
    if( ! in_cell(i) && in_cell(f)) {
        // Flip a coin to decide whether or not to diffuse into the cell
    }

    // Handle the case where water diffuses out of the cell
    if(in_cell(i) && ! in_cell(f)) {
        // Flip a coin to decide whether or not to diffuse out of the cell
    }
}

/**
 * Compute reflection off MNPs trivially - that is, by looping over all MNPs
 * and checking the distance norm to them. Note that this function doesn't
 * actually apply the reflection.
 *
 * Returns true if a reflection needs to occur and false otherwise.
 */
__device__ bool mnpReflection(water_info* w, int mnp_count, MNP_info *mnps) {
    bool reflect = false;
    for(int i = 0; i < mnp_count; i++) {
        double dx = mnps[i].x - w->x;
        double dy = mnps[i].y - w->y;
        double dz = mnps[i].z - w->z;
        double r = mnps[i].r;
        if(NORMSQ(dx, dy, dz) < r * r) {
            reflect = true;
        }
    }
    return reflect;
}

/**
 * Memory reduction: Use the shared memory in each block to sum up the
 * magnetizations for all water molecules processed by the threads in the block.
 * Design pattern from CUDA by Design (c).
 *
 * @param cache A pointer to the shared memory cache for this block
 * @param t     Pointer to global memory to store the target sum
 */
__device__ void sumMag(double* cache, double* t) {
    int tid = threadIdx.x;
    int i = blockDim.x / 2;

    while(i != 0) {
        if(tid < i) {
            cache[tid] += cache[tid + i];
        }
        __syncthreads();
    }

    // Copy the sum back into global memory
    if(tid == 0) {
        *t = cache[0];
    }
}

__device__ double dipole_field(double dx, double dy, double dz, double M)
{
    double divisor = pow(NORMSQ(dx, dy, dz), 2.5);
    return M * 1e11 * (2*dz*dz - dx*dx - dy*dy) / divisor;
}

/**
 * Causes a water molecule to accumulate phase according to the magnetic
 * field that it experiences. Currently implemented to simply sum up
 * the field contributions of all MNPs in the volume - the trivial
 * implementation.
 */
__device__ void accumulatePhase(water_info* w, int mnp_count, MNP_info* mnps) {
    double B = 0;
    for(int i = 0; i < mnp_count; i++) {
        double dx = mnps[i].x - w->x;
        double dy = mnps[i].y - w->y;
        double dz = mnps[i].z - w->z;
    }
    w->phase += B * 2 * M_PI * g * tau * 1e-3;
}

/**
 * Constrains the water molecules to lie within the bounds of the simulation.
 * Right now, it is implemented to NOT track the number of boundary crossings
 * for each water molecule.
 */
__device__ void boundary_conditions(water_info* w) {
    w->x %= bound;
    w->y %= bound;
    w->z %= bound;
}

/**
 * Kernel that actually performs the diffusing water simulation
 */
__global__ void waterSimulate(GPUData d) {
    __shared__ double mags[threads_per_block];
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    initRandomState(tid, d.states, )

    for(int i = 0; i < t; i++) {
        if(tid < num_water) {
            water_info *w = d->waters[tid];
            for(int i = 0; i < t; i++) {
              // TODO: Take a random step in a direction
              // TODO: Check for boundary condition crossing
              boundary_conditions(w);
              // TODO: Check for cell reflection
              // TODO: Check for MNP reflection
              // Accumulate phase
              accumulatePhase(w);

              // TODO: Apply a flip at the Carr-Purcell time

              // Store the magnetizations in a cache
              mags[threadIdx.x] = cos(w->phase);
              __syncthreads();
              // Sum up the net magnetizations, store in the target
              sumMag(tid, mags, d.magnetizations + i * num_blocks + blockIdx.x);
            }
        }
    }
}

double* gpuSimulate(vector<MNP_info> * mnps, water_info* w) {
    GPUData d = initializeGPU(mnps, w);
    waterSimulate<<<num_blocks, threads_per_block>>>(d);
    return finalizeGPU(d);
}
