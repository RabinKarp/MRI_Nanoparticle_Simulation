#include "hip/hip_runtime.h"
#define HIGH           5000000
#define MAX_MNPS       1000
#define M_PI           3.14159265358979323846

#include "utilities/utilities.h"
#include "cuda_sim.h"

#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <string>

#include <vector>
#include <fstream>
#include <thread>
#include <iostream>
#include "math.h"
#include "parameters.h"
#include "fcc_diffusion.h"
#include "gpu_random.h"
#include "octree.h"

using namespace std;

struct t_rands {
    double *uniform;
    double *norm;
};

const int num_blocks = (num_water + threads_per_block - 1) / threads_per_block;

const double g = 42.5781e6;             // gyromagnetic ratio in MHz/T
const double pInt = 1e-3;
const int pfreq = (int)(pInt/tau);      // print net magnetization every 1us

#define num_uniform_doubles 4 // # of uniform doubles per water per tstep
#define num_normal_doubles 2  // # of normal  doubles per water per tstep

//==============================================================================
// Octree-related functions
//==============================================================================

__device__ double dipole_field(double dx, double dy, double dz, double M, GPUData &d)
{
    double sqDist = NORMSQ(dx, dy, dz);
    double divisor = sqDist * sqDist * sqrt(sqDist);
    return (sqDist > d.cell_r * d.cell_r) * M * 1e11 * (2*dz*dz - dx*dx - dy*dy) / divisor;
}

__device__ uint64_t morton_code(int depth, double &x, double &y, double &z, GPUData &d) {
    uint64_t size = 1 << (depth);
    uint32_t idx_x = floor(x / d.bound * size);
    uint32_t idx_y = floor(y / d.bound * size);
    uint32_t idx_z = floor(z / d.bound * size);
    uint64_t answer = 0;
    // start by shifting the third byte, since we only look @ the first 21 bits
    if (depth > 16)
    {
        answer |=   d.morton_z[(idx_z >> 16) & 0xFF ] |
                    d.morton_y[(idx_y >> 16) & 0xFF ] |
                    d.morton_x[(idx_x >> 16) & 0xFF ];
        answer <<= 24;
    }

    // shift second byte
    if (depth > 8)
    {
        answer |=   d.morton_z[(idx_z >> 8) & 0xFF ] |
                    d.morton_y[(idx_y >> 8) & 0xFF ] |
                    d.morton_x[(idx_x >> 8) & 0xFF ];
        answer <<= 24;
    }

    // shift first byte
    answer |=   d.morton_z[(idx_z) & 0xFF ] |
                d.morton_y[(idx_y) & 0xFF ] |
                d.morton_x[(idx_x) & 0xFF ];

    return answer;
}

__device__ gpu_node** find_tree(double wx, double wy, double wz, GPUData &d)
{
    return d.tree + morton_code(d.min_depth, wx, wy, wz, d);
}

/*
 * Helper function to find the child index of a parent node at depth d that
 * holds the water molecule at (wx, wy, wz). This is done by taking the Morton
 * code of (wx, wy, wz) at depth d + 1 and returning the last 3 bits, which
 * would encode one additional level of depth.
 */
__device__ unsigned find_child(double wx, double wy, double wz, int d, GPUData &data)
{
    return morton_code(d + 1, wx, wy, wz, data) & 7;
}

// TODO: Check this function!
__device__ gpu_node* find_node(gpu_node *n, double wx, double wy, double wz, int d, GPUData &data) {
    // base case -- node is a leaf
    if (n->mc >> 63)
        return n;

    // otherwise, navigate to the appropriate child and recurse
    unsigned child_no = find_child(wx, wy, wz, d, data);
    return find_node(n + n->child[child_no].idx, wx, wy, wz, d + 1, data);
}

__device__ gpu_node* get_voxel(water_info *w, GPUData &d) {
    double wx = w->x, wy = w->y, wz = w->z;
    return find_node(*(find_tree(wx, wy, wz, d)), wx, wy, wz, d.min_depth, d);
}

/**
 * Returns the B field at the location of a particular water molecule
 */
__device__ double get_field(water_info *w, gpu_node* leaf, GPUData &d) {
    double wx = w->x, wy = w->y, wz = w->z;

    uint64_t depth = 0, mc = (leaf->mc << 1) >> 1;
    while (mc >>= 3) depth++;

    // use Morton code's depth to find child index to find value of B to return
    unsigned child_no = find_child(wx, wy, wz, depth, d);
    double B = (double)leaf->child[child_no].B;

    // add in contributions from resident MNPs zeroed out during construction
    for(int i = 0; i < leaf->numResidents; i++) {
        MNP_info *np = leaf->resident + i;
        B += dipole_field(wx - np->x, wy - np->y, wz - np->z, np->M, d);
    }

    return B;
}

/**
 * Initialize a GPU verison of the octree from the CPU version.
 */
void initOctree(Octree *oct, GPUData &d) {
    // Initialize octree parameters
    d.min_depth = oct->min_depth;
    d.max_depth = oct->max_depth;
    d.addresses = new std::vector<void*>();

    vector<oct_node> *vec_nodes = oct->space;
    vector<MNP_info> &vec_mnps = *(oct->mnps);

    int arr_size = (int) pow(8, d.min_depth);
    d.arr_size = arr_size;


    gpu_node** localTree = new gpu_node*[arr_size];
    gpu_node** localPointers = new gpu_node*[arr_size];
    d.sizes = new int[arr_size];

    bool* checked = new bool[vec_mnps.size()];

    for(int i = 0; i < arr_size; i++) {
        if(vec_nodes + i) {
          vector<oct_node> &current = vec_nodes[i];
          localTree[i] = new gpu_node[current.size()];

          for(int j = 0; j < current.size(); j++) {
              localTree[i][j].mc = current[j].mc;

              for(int k = 0; k < 8; k++) {
                  localTree[i][j].child[k] = current[j].child[k];
              }

              if(current[j].resident) {
                  localTree[i][j].numResidents = current[j].resident->size();

                  // This will become a device pointer
                  localTree[i][j].resident = (MNP_info*) cudaAllocate(sizeof(MNP_info) * localTree[i][j].numResidents);
                  d.addresses->push_back((void*) localTree[i][j].resident);
                  // Copy MNPs to device
                  copyToDevice((void *) localTree[i][j].resident,
                      (void*) current[j].resident->data(),
                      sizeof(MNP_info) * localTree[i][j].numResidents);
              }
              else {
                  localTree[i][j].numResidents = 0;
                  localTree[i][j].resident = nullptr;
              }
          }
          // Store the subtree size locally
          d.sizes[i] = current.size();
          // Now copy the entire subtree to the device, storing in the local pointers struct
          localPointers[i] = (gpu_node *) cudaAllocate(sizeof(gpu_node) * current.size());
          d.addresses->push_back((void*) localPointers[i]);
          copyToDevice((void*)localPointers[i], (void*) localTree[i], sizeof(gpu_node) * current.size());
        }

        // Now copy the entire tree into memory
        d.tree = (gpu_node**) cudaAllocate(sizeof(gpu_node**) * arr_size);
        copyToDevice((void*) d.tree, (void*) localPointers, sizeof(gpu_node**) * arr_size);
    }

    d.morton_x = (uint32_t*) cudaAllocate(256 * sizeof(uint32_t));
    d.morton_y = (uint32_t*) cudaAllocate(256 * sizeof(uint32_t));
    d.morton_z = (uint32_t*) cudaAllocate(256 * sizeof(uint32_t));

    copyToDevice((void*) d.morton_x,(void*) morton_x, 256 * sizeof(uint32_t));
    copyToDevice((void*) d.morton_y,(void*) morton_y, 256 * sizeof(uint32_t));
    copyToDevice((void*) d.morton_z,(void*) morton_z, 256 * sizeof(uint32_t));

    for(int i = 0; i < arr_size; i++) {
        delete[] localTree[i];
    }
}

void destroyTree(GPUData &d) {
    // TODO: Fix memory cleanup here

    hipFree(d.tree);

    for(std::vector<void*>::iterator it = d.addresses->begin(); it != d.addresses->end(); it++) {
        hipFree(*it);
    }

    cout << "Tree freed!" << endl;

    // Free the morton code arrays
    hipFree(d.morton_x);
    hipFree(d.morton_y);
    hipFree(d.morton_z);

    // Host pointer, can call delete
    delete[] d.sizes;
    delete d.addresses;
}

//==============================================================================
// GPU setup functions
//==============================================================================

void finalizeGPU(GPUData &d) {
    hipFree(d.waters);
    hipFree(d.flags);
    hipFree(d.uniform_doubles);
    hipFree(d.normal_doubles);
    hipFree(d.magnetizations);
    hipFree(d.lattice);
    hipFree(d.time);

    destroyTree(d);
}

void setParameters(GPUData &d) {
    // Initialize constants for the GPU
    d.in_stdev = sqrt(pi * D_cell * tau);
    d.out_stdev = sqrt(pi * D_extra * tau);

    d.reflectIO = 1 - sqrt(tau / (6*D_cell)) * 4 * P_expr;
    d.reflectOI = 1 - ((1 - d.reflectIO) * sqrt(D_cell/D_extra));
    d.tcp = tcp;

    d.num_cells = num_cells;
    d.num_waters = num_water;
    d.timesteps = sprintSteps;
    d.cell_r = cell_r;
    d.bound = bound;
    d.nBlocks = num_blocks;
    d.g = g;
    d.tau = tau;
    d.bound = bound;
    d.pfreq = pfreq;
    d.hashDim = hashDim;
    d.phase_stdev = phase_stdev;
}

//==============================================================================
// Simulation functions
//==============================================================================

__device__ void updateNearest(water_info *w, GPUData &d) {
    double cubeLength = d.bound / d.hashDim;
    int x_idx = w->x / cubeLength;
    int y_idx = w->y / cubeLength;
    int z_idx = w->z / cubeLength;

    int* nearest =
        d.lookupTable[z_idx * d.hashDim * d.hashDim
        + y_idx * d.hashDim
        + x_idx];

    double cDist = d.bound * d.bound * 3;
    int cIndex = -1;
    while(*nearest != -1) {
        double dx = d.lattice[*nearest].x - w->x;
        double dy = d.lattice[*nearest].y - w->y;
        double dz = d.lattice[*nearest].z - w->z;

        double dist = NORMSQ(dx, dy, dz);
        if(NORMSQ(dx, dy, dz) < cDist) {
            cDist = dist;
            cIndex = *nearest;
        }
        nearest++;
    }

    w->in_cell = (cDist < d.cell_r * d.cell_r);
    w->nearest = cIndex;
}

__device__ bool cell_reflect(water_info *i, water_info *f, int tStep, GPUData &d) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    double coin = d.uniform_doubles[tStep * d.num_waters * 4 + tid * 4 + 3];
    bool flip = (i->in_cell && (! f->in_cell) && coin < d.reflectIO)
                    || ((! i->in_cell) && f->in_cell && coin < d.reflectOI);
    return flip;
}

__device__ bool mnp_reflect(water_info *w, MNP_info *mnp, int num_mnps, GPUData &d) {
    bool retValue = false;

    for(int i = 0; i < num_mnps; i++) {
        MNP_info* m = mnp + i;
        double dx = m->x - w->x;
        double dy = m->y - w->y;
        double dz = m->z - w->z;

        if(NORMSQ(dx, dy, dz) < (m->r * m->r))
            retValue = true;
    }

    return retValue;
}

__device__ water_info rand_displacement(water_info *w, t_rands *r_nums, GPUData &d) {
    water_info disp;
    double norm = *(r_nums->norm);

    disp.x = *(r_nums->uniform + 0) * 2 - 1.0;
    disp.y = *(r_nums->uniform + 1) * 2 - 1.0;
    disp.z = *(r_nums->uniform + 2) * 2 - 1.0;

    if(w->in_cell) {
        norm *= d.in_stdev;
    }
    else {
        norm *= d.out_stdev;
    }

    double nConstant = norm / sqrt(NORMSQ(disp.x, disp.y, disp.z));

    disp.x *= nConstant;
    disp.y *= nConstant;
    disp.z *= nConstant;

    return disp;
}

__device__ void boundary_conditions(water_info *w, GPUData &d) {
    w->x = fmod(w->x + d.bound, d.bound);
    w->y = fmod(w->y + d.bound, d.bound);
    w->z = fmod(w->z + d.bound, d.bound);
}

__device__ void accumulatePhase(water_info *w, gpu_node* voxel, t_rands *r_nums, GPUData &d) {
    double B = get_field(w, voxel, d);
    double nD = * (r_nums->norm + 1);

    // If inside a cell, add a random phase kick.
    w->phase += (w->in_cell) * nD * d.phase_stdev;
    w->phase += B * 2 * M_PI * d.g * d.tau * 1e-3;
}
// END PHASE ACCUMULATION FUNCTIONS

__device__ void sumMagnetizations(double* input, int timepoint, GPUData &d) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    int i = blockDim.x/2;
    __syncthreads();
    while(i != 0) {
        if( (cacheIndex < i) && ((tid + i) < d.num_waters) ) {
            input[cacheIndex] += input[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }
    if(cacheIndex == 0) {
        d.magnetizations[timepoint * d.nBlocks + blockIdx.x] = input[0];
    }
}

__global__ void simulateWaters(GPUData d)  {
    __shared__ double mags[threads_per_block];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int startTime = *d.time;
    water_info w;
    gpu_node *voxel;

    int x = 0;

    if(tid < d.num_waters) {
        // Copy water to chip memory
        w = d.waters[tid];
        d.flags[tid] = -1;
        updateNearest(&w, d);
    }

    struct t_rands r_nums;
    r_nums.uniform = d.uniform_doubles + tid * num_uniform_doubles;
    r_nums.norm = d.normal_doubles + tid * num_normal_doubles;

    for(int i = 0; i < d.timesteps; i++) {

        if(tid < d.num_waters) {
            x++;
            water_info init = w;

            water_info disp = rand_displacement(&w, &r_nums, d);
            w.x += disp.x;
            w.y += disp.y;
            w.z += disp.z;
            boundary_conditions(&w, d);
            updateNearest(&w, d);
            voxel = get_voxel(&w, d);

            // Check cell boundary / MNP reflection

            if(cell_reflect(&init, &w, i, d)) {
                w = init;
            }

            accumulatePhase(&w, voxel, &r_nums, d);

            if(((startTime + i) % (2 * d.tcp)) == d.tcp) {
                w.phase *= -1;
            }

            // If we need to do a reduction, copy to shared memory
            if((startTime + i) % d.pfreq == 0)
                mags[threadIdx.x] = cos(w.phase);
        }
        // Perform a memory reduction

        if((startTime + i) % d.pfreq == 0) {
            sumMagnetizations(mags, i / d.pfreq, d);
        }
        r_nums.uniform += d.num_waters * num_uniform_doubles;
        r_nums.norm += d.num_waters * num_normal_doubles;
    }

    __syncthreads();
    if(tid == 0) {
        *d.time += d.timesteps;
    }

    // Copy the water molecule back to global memory
    if(tid < d.num_waters) {
        d.flags[tid] = x;
        d.waters[tid] = w;
    }
}

void cpyLookupDevice(int **sourceTable, GPUData &d) {
    d.localLookup = new int*[hashDim * hashDim * hashDim];

    for(int i = 0; i < hashDim * hashDim * hashDim; i++) {
        d.localLookup[i] = (int *) cudaAllocate(maxNeighbors * sizeof(int));
        copyToDevice((void *) d.localLookup[i], (void *) sourceTable[i],
            maxNeighbors * sizeof(int));
    }
    d.lookupTable = (int**) cudaAllocate(hashDim * hashDim * hashDim * sizeof(int**));
    copyToDevice((void *) d.lookupTable, d.localLookup,
        hashDim * hashDim * hashDim * sizeof(int*));
}

void destroyLookupDevice(GPUData &d) {
    for(int i = 0; i < hashDim * hashDim * hashDim; i++) {
        hipFree(d.localLookup[i]);
    }
    hipFree(d.lookupTable);
    delete[] d.localLookup;
}

void simulateWaters(std::string filename) {
    cout << "Starting GPU Simulation..." << endl;
    cout << "Printing to: " << filename << endl;
    ofstream fout(filename);

    hipEvent_t start, stop;

    // Initialize PRNG seed for MNPs and waters
    std::random_device rd;
    XORShift<uint64_t> gen(time(NULL) + rd());

    // The simulation has 3 distinct components: the lattice, the water
    // molecules, and the nanoparticles

    FCC lattice(D_cell, D_extra, P_expr, gen);
    vector<MNP_info> *mnps = lattice.init_mnps();
    water_info *waters = lattice.init_molecules(num_water,gen);
    Triple* linLattice = lattice.linearLattice();

    // Initialize the octree
    double max_product = 2e-6, max_g = 5, min_g = .002;
    uint64_t sTime = time(NULL);
    Octree tree(max_product, max_g, min_g, gen, mnps);
    uint64_t eTime = time(NULL) - sTime;
    std::cout << "Octree took " << eTime / 60 << ":";
    if (eTime % 60 < 10) std::cout << "0";
    std::cout << eTime % 60 << " to build." << std::endl << std::endl;

    GPUData d;
    setParameters(d);
    d.num_mnps = mnps->size();
    initOctree(&tree, d);

    cout << "Allocated GPU Octree!" << endl;
    int totalUniform =  num_uniform_doubles * num_water * sprintSteps;
    int totalNormal = num_normal_doubles * num_water * sprintSteps;
    int initTime = 0;

    // Allocations: Perform all allocations here
    HANDLE_ERROR(hipMalloc((void **) &(d.waters),
        num_water * sizeof(water_info)));
    HANDLE_ERROR(hipMalloc((void **) &(d.flags),
        num_water * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **) &d.uniform_doubles,
        totalUniform*sizeof(double)));
    HANDLE_ERROR(hipMalloc((void **) &d.normal_doubles,
        totalNormal*sizeof(double)));
    HANDLE_ERROR(hipMalloc((void **) &d.time,
        sizeof(int)));

    d.lattice = (Triple*) cudaAllocate(sizeof(Triple) * num_cells);
    copyToDevice(d.lattice, linLattice, num_cells * sizeof(Triple));
    cpyLookupDevice(lattice.lookupTable, d);

    // Allocate the target array
    HANDLE_ERROR(hipMalloc((void **) &(d.magnetizations),
        num_blocks * (t / pfreq) * sizeof(double)));

    // Initialize performance timers
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    // Perform all memory copies here
    HANDLE_ERROR(hipMemcpy(d.waters, waters,
        sizeof(water_info) * num_water,
        hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d.time, &initTime,
        sizeof(int),
        hipMemcpyHostToDevice));

    int flags[num_water];
    double *magnetizations = new double[num_blocks * (t / pfreq)]; // Local magnetization target

    cout << "Kernel prepped!" << endl;

    // Run the kernel in sprints due to memory limits and timeout issues
    double time = 0;
    fout << time << "," << num_water << endl;
    for(int i = 0; i < (t / sprintSteps); i++) {
        cout << "Starting sprint " << (i+1) << "." << endl;
        getUniformDoubles(totalUniform, d.uniform_doubles);
        getNormalDoubles(totalNormal, d.normal_doubles);

        HANDLE_ERROR(hipEventRecord(start, 0));
        simulateWaters<<<num_blocks, threads_per_block>>>(d);

        HANDLE_ERROR(hipEventRecord(stop, 0));
        HANDLE_ERROR(hipEventSynchronize(stop));

        float elapsedTime;
        HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

        cout << "Kernel execution complete! Elapsed time: "
            << elapsedTime << " ms" << endl;

        // Copy back the array of flags to catch any errors
        HANDLE_ERROR(hipMemcpy(flags, d.flags,
            sizeof(int) * num_water,
            hipMemcpyDeviceToHost));

        bool success = true;
        for(int i = 0; i < num_water; i++) {
            if(flags[i] != sprintSteps)
                success = false;
        }
        cout << "Success State: " << success << endl << "===========" << endl;

        // Copy back the array of magnetizations
        HANDLE_ERROR(hipMemcpy(magnetizations, d.magnetizations,
            num_blocks * (t / pfreq) * sizeof(double),
            hipMemcpyDeviceToHost));

        for(int j = 0; j < sprintSteps / pfreq; j++) {
            double magSum = 0;
            for(int k = 0; k < num_blocks; k++) {
                magSum += magnetizations[j * num_blocks + k];
            }
            time += pInt;
            fout << time << "," << magSum << endl;
        }
    }

    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));
    destroyLookupDevice(d);
    finalizeGPU(d);

    delete[] linLattice;
    delete[] waters;
    delete[] magnetizations;
    delete mnps;
    fout.close();
}
