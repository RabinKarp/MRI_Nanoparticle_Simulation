#include "hip/hip_runtime.h"
#define HIGH           5000000
#define MAX_MNPS       1000
#define M_PI           3.14159265358979323846

#include "cuda_helpers.h"

#include <stdio.h>
#include <time.h>
#include <stdlib.h>

#include <vector>
#include <fstream>
#include <thread>
#include <iostream>
#include "math.h"
#include "parameters.h"
#include "fcc_diffusion.h"
#include "gpu_random.h"
#include "octree.h"

using namespace std;
/**
 *  nvcc cuda_test.cu fcc_diffusion.cpp rand_walk.cpp octree.cpp -arch=sm_61 -lcurand -ccbin "
 * C:\Program Files (x86)\Microsoft Visual Studio 14.0\VC\bin\amd64_x86"
 */

 /*
  * nvcc cuda_test.cu fcc_diffusion.cpp rand_walk.cpp octree.cpp -arch=s
  * m_61 -lcurand -ccbin "C:\Program Files (x86)\Microsoft Visual Studio 14.0\VC\bin\amd64_x86"
  */

#define threads_per_block 32
const int num_blocks = (num_water + threads_per_block - 1) / threads_per_block;

const double g = 42.5781e6;             // gyromagnetic ratio in MHz/T
const int pfreq = (int)(1e-3/tau);      // print net magnetization every 1us

// Each kernel execution handles AT MOST this many timesteps
const int sprintSteps = 10000;
const int num_uniform_doubles = 4; // # of uniform doubles per water per tstep
const int num_normal_doubles = 1;  // # of normal  doubles per water per tstep

__constant__ Triple dev_lattice[num_cells];

inline void* cudaAllocate(long long int size) {
    void *ptr;
    HANDLE_ERROR(hipMalloc((void **) &ptr, size));
    return ptr;
}

inline void copyToDevice(void* dest, void* source, long long int size) {
    HANDLE_ERROR(hipMemcpy(dest, source,
        size,
        hipMemcpyHostToDevice));
}

inline void copyToHost(void* dest, void* source, long long int size) {
    HANDLE_ERROR(hipMemcpy(dest, source,
        size,
        hipMemcpyDeviceToHost));
}

//==============================================================================

__device__ double dipole_field(double dx, double dy, double dz, double M)
{
    double divisor = pow(NORMSQ(dx, dy, dz), 2.5);
    return M * 1e11 * (2*dz*dz - dx*dx - dy*dy) / divisor;
}

__device__ uint64_t morton_code(int depth, double x, double y, double z, GPUData &d) {

    double size = pow(2.0, depth);
    uint32_t idx_x = floor(x / d.bound * size);
    uint32_t idx_y = floor(y / d.bound * size);
    uint32_t idx_z = floor(z / d.bound * size);
    uint64_t answer = 0;
    // start by shifting the third byte, since we only look @ the first 21 bits
    if (depth > 16)
    {
        answer |=   d.morton_z[(idx_z >> 16) & 0xFF ] |
                    d.morton_y[(idx_y >> 16) & 0xFF ] |
                    d.morton_x[(idx_x >> 16) & 0xFF ];
        answer <<= 24;
    }

    // shift second byte
    if (depth > 8)
    {
        answer |=   d.morton_z[(idx_z >> 8) & 0xFF ] |
                    d.morton_y[(idx_y >> 8) & 0xFF ] |
                    d.morton_x[(idx_x >> 8) & 0xFF ];
        answer <<= 24;
    }

    // shift first byte
    answer |=   d.morton_z[(idx_z) & 0xFF ] |
                d.morton_y[(idx_y) & 0xFF ] |
                d.morton_x[(idx_x) & 0xFF ];

    return answer;
}

__device__ gpu_node** find_tree(double wx, double wy, double wz, GPUData &d)
{
    return d.tree + morton_code(d.min_depth, wx, wy, wz, d);
}

/*
 * Helper function to find the child index of a parent node at depth d that
 * holds the water molecule at (wx, wy, wz). This is done by taking the Morton
 * code of (wx, wy, wz) at depth d + 1 and returning the last 3 bits, which
 * would encode one additional level of depth.
 */
__device__ unsigned find_child(double wx, double wy, double wz, int d, GPUData &data)
{
    return morton_code(d + 1, wx, wy, wz, data) & 7;
}

// TODO: Check this function!
__device__ gpu_node* find_node(gpu_node *n, double wx, double wy, double wz, int d, GPUData &data) {
    // base case -- node is a leaf
    if (n->mc >> 63)
        return n;

    // otherwise, navigate to the appropriate child and recurse
    unsigned child_no = find_child(wx, wy, wz, d, data);
    return find_node(n + n->child[child_no].idx, wx, wy, wz, d + 1, data);
}

__device__ gpu_node* get_voxel(water_info *w, GPUData &d) {
    double wx = w->x, wy = w->y, wz = w->z;
    return find_node(*(find_tree(wx, wy, wz, d)), wx, wy, wz, d.min_depth, d);
}

/**
 * Returns the B field at the location of a particular water molecule
 */
__device__ double get_field(water_info *w, gpu_node* leaf, GPUData &d) {
    double wx = w->x, wy = w->y, wz = w->z;

    uint64_t depth = 0, mc = (leaf->mc << 1) >> 1;
    while (mc >>= 3) depth++;

    // use Morton code's depth to find child index to find value of B to return
    unsigned child_no = find_child(wx, wy, wz, depth, d);
    double B = (double)leaf->child[child_no].B;

    // add in contributions from resident MNPs zeroed out during construction
    for(int i = 0; i < leaf->numResidents; i++) {
        MNP_info *np = leaf->resident + i;
        B += dipole_field(wx - np->x, wy - np->y, wz - np->z, np->M);
    }

    return B;
}

/**
 * Initialize a GPU verison of the octree from the CPU version.
 */
void initOctree(Octree *oct, GPUData &d) {
    // Initialize octree parameters
    d.min_depth = oct->min_depth;
    d.max_depth = oct->max_depth;
    d.addresses = new std::vector<void*>();

    vector<oct_node> *vec_nodes = oct->space;
    vector<MNP_info> &vec_mnps = *(oct->mnps);

    int arr_size = (int) pow(8, d.min_depth);
    d.arr_size = arr_size;


    gpu_node** localTree = new gpu_node*[arr_size];
    gpu_node** localPointers = new gpu_node*[arr_size];
    d.sizes = new int[arr_size];

    bool* checked = new bool[vec_mnps.size()];

    for(int i = 0; i < arr_size; i++) {
        if(vec_nodes + i) {
          vector<oct_node> &current = vec_nodes[i];
          localTree[i] = new gpu_node[current.size()];

          for(int j = 0; j < current.size(); j++) {
              localTree[i][j].mc = current[j].mc;

              for(int k = 0; k < 8; k++) {
                  localTree[i][j].child[k] = current[j].child[k];
              }

              if(current[j].resident) {
                  localTree[i][j].numResidents = current[j].resident->size();

                  // This will become a device pointer
                  localTree[i][j].resident = (MNP_info*) cudaAllocate(sizeof(MNP_info) * localTree[i][j].numResidents);
                  d.addresses->push_back((void*) localTree[i][j].resident);
                  // Copy MNPs to device
                  copyToDevice((void *) localTree[i][j].resident,
                      (void*) current[j].resident->data(),
                      sizeof(MNP_info) * localTree[i][j].numResidents);
              }
              else {
                  localTree[i][j].numResidents = 0;
                  localTree[i][j].resident = nullptr;
              }
          }
          // Store the subtree size locally
          d.sizes[i] = current.size();
          // Now copy the entire subtree to the device, storing in the local pointers struct
          localPointers[i] = (gpu_node *) cudaAllocate(sizeof(gpu_node) * current.size());
          d.addresses->push_back((void*) localPointers[i]);
          copyToDevice((void*)localPointers[i], (void*) localTree[i], sizeof(gpu_node) * current.size());
        }

        // Now copy the entire tree into memory
        d.tree = (gpu_node**) cudaAllocate(sizeof(gpu_node**) * arr_size);
        copyToDevice((void*) d.tree, (void*) localPointers, sizeof(gpu_node**) * arr_size);
    }

    d.morton_x = (uint32_t*) cudaAllocate(256 * sizeof(uint32_t));
    d.morton_y = (uint32_t*) cudaAllocate(256 * sizeof(uint32_t));
    d.morton_z = (uint32_t*) cudaAllocate(256 * sizeof(uint32_t));

    copyToDevice((void*) d.morton_x,(void*) morton_x, 256 * sizeof(uint32_t));
    copyToDevice((void*) d.morton_y,(void*) morton_y, 256 * sizeof(uint32_t));
    copyToDevice((void*) d.morton_z,(void*) morton_z, 256 * sizeof(uint32_t));

    for(int i = 0; i < arr_size; i++) {
        delete[] localTree[i];
    }
}

void destroyTree(GPUData &d) {
    // TODO: Fix memory cleanup here

    hipFree(d.tree);

    for(std::vector<void*>::iterator it = d.addresses->begin(); it != d.addresses->end(); it++) {
        hipFree(*it);
    }

    cout << "Tree freed!" << endl;

    // Free the morton code arrays
    hipFree(d.morton_x);
    hipFree(d.morton_y);
    hipFree(d.morton_z);

    // Host pointer, can call delete
    delete[] d.sizes;
    delete d.addresses;
}
//==============================================================================

void finalizeGPU(GPUData &d) {
    hipFree(d.waters);
    hipFree(d.flags);
    hipFree(d.uniform_doubles);
    hipFree(d.normal_doubles);
    hipFree(d.magnetizations);
    hipFree(d.lattice);
    hipFree(d.time);

    destroyTree(d);
}

void setParameters(GPUData &d) {
    // Initialize constants for the GPU
    d.in_stdev = sqrt(pi * D_cell * tau);
    d.out_stdev = sqrt(pi * D_extra * tau);

    d.reflectIO = 1 - sqrt(tau / (6*D_cell)) * 4 * P_expr;
    d.reflectOI = 1 - ((1 - d.reflectIO) * sqrt(D_cell/D_extra));
    d.tcp = tcp;

    d.num_cells = num_cells;
    d.num_waters = num_water;
    d.timesteps = sprintSteps;
    d.cell_r = cell_r;
    d.bound = bound;
    d.nBlocks = num_blocks;
    d.g = g;
    d.tau = tau;
    d.bound = bound;
    d.pfreq = pfreq;
    d.hashDim = hashDim;
}

/**
 * Trivial implementation of nearest cell finder
 */
__device__ void updateNearest(water_info *w, GPUData &d) {
    double cubeLength = d.bound / d.hashDim;
    int x_idx = w->x / cubeLength;
    int y_idx = w->y / cubeLength;
    int z_idx = w->z / cubeLength;

    int* nearest =
        d.lookupTable[z_idx * d.hashDim * d.hashDim
        + y_idx * d.hashDim
        + x_idx];

    double cDist = d.bound * d.bound * 3;
    int cIndex = -1;
    while(*nearest != -1) {
        double dx = d.lattice[*nearest].x - w->x;
        double dy = d.lattice[*nearest].y - w->y;
        double dz = d.lattice[*nearest].z - w->z;

        double dist = NORMSQ(dx, dy, dz);
        if(NORMSQ(dx, dy, dz) < cDist) {
            cDist = dist;
            cIndex = *nearest;
        }
        nearest++;
    }

    w->in_cell = (cDist < d.cell_r * d.cell_r);
    w->nearest = cIndex;
}

__device__ bool cell_reflect(water_info *i, water_info *f, int tStep, GPUData &d) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    double coin = d.uniform_doubles[tStep * d.num_waters * 4 + tid * 4 + 3];
    bool flip = (i->in_cell && (! f->in_cell) && coin < d.reflectIO)
                    || ((! i->in_cell) && f->in_cell && coin < d.reflectOI);
    return flip;
}

__device__ bool mnp_reflect(water_info *w, MNP_info *mnp, int num_mnps, GPUData &d) {
    bool retValue = false;

    for(int i = 0; i < num_mnps; i++) {
        MNP_info* m = mnp + i;
        double dx = m->x - w->x;
        double dy = m->y - w->y;
        double dz = m->z - w->z;

        if(NORMSQ(dx, dy, dz) < (m->r * m->r))
            retValue = true;
    }

    return retValue;
}

__device__ water_info rand_displacement(int tid, int tStep, water_info *w, GPUData &d) {
    water_info disp;
    double norm = d.normal_doubles[tStep * d.num_waters + tid];
    int baseU = tStep * d.num_waters * 4 + tid * 4;

    disp.x = d.uniform_doubles[baseU] * 2 - 1.0;
    disp.y = d.uniform_doubles[baseU + 1] * 2 - 1.0;
    disp.z = d.uniform_doubles[baseU + 2] * 2 - 1.0;

    if(w->in_cell) {
        norm *= d.in_stdev;
    }
    else {
        norm *= d.out_stdev;
    }

    double nConstant = norm / sqrt(NORMSQ(disp.x, disp.y, disp.z));

    disp.x *= nConstant;
    disp.y *= nConstant;
    disp.z *= nConstant;

    return disp;
}

__device__ void boundary_conditions(water_info *w, GPUData &d) {
    w->x = fmod(w->x + d.bound, d.bound);
    w->y = fmod(w->y + d.bound, d.bound);
    w->z = fmod(w->z + d.bound, d.bound);
}

__device__ void accumulatePhase(water_info *w, gpu_node* voxel, GPUData &d) {
    double B = get_field(w, voxel, d);
    w->phase += B * 2 * M_PI * d.g * d.tau * 1e-3;
}
// END PHASE ACCUMULATION FUNCTIONS

__device__ void sumMagnetizations(double *input, int timepoint, GPUData &d) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int cacheIndex = threadIdx.x;
    int i = blockDim.x/2;
    __syncthreads();
    while(i != 0) {
        if( (cacheIndex < i) && ((tid + i) < d.num_waters) ) {
            input[cacheIndex] += input[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }
    if(cacheIndex == 0) {
        d.magnetizations[timepoint * d.nBlocks + blockIdx.x] = input[0];
    }
}

__global__ void simulateWaters(GPUData d)  {
    __shared__ double mags[threads_per_block];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    int startTime = *d.time;
    water_info w;
    gpu_node *voxel;


    int x = 0;

    if(tid < d.num_waters) {
        // Copy water to chip memory
        w = d.waters[tid];
        d.flags[tid] = -1;
        updateNearest(&w, d);
    }

    for(int i = 0; i < d.timesteps; i++) {

        if(tid < d.num_waters) {
            x++;
            water_info init = w;

            water_info disp = rand_displacement(tid, i, &w, d);
            w.x += disp.x;
            w.y += disp.y;
            w.z += disp.z;
            boundary_conditions(&w, d);
            updateNearest(&w, d);
            voxel = get_voxel(&w, d);

            // Check cell boundary / MNP reflection

            if(cell_reflect(&init, &w, i, d) || mnp_reflect(&w, voxel->resident, voxel->numResidents,d)) {
                w = init;
            }

            accumulatePhase(&w, voxel, d);

            if(((startTime + i) % (2 * d.tcp)) == d.tcp) {
                w.phase *= -1;
            }

            // If we need to do a reduction, copy to shared memory
            if((startTime + i) % d.pfreq == 0)
                mags[threadIdx.x] = cos(w.phase);
        }
        // Perform a memory reduction

        if((startTime + i) % d.pfreq == 0) {
            sumMagnetizations(mags, i / d.pfreq, d);
        }
    }

    __syncthreads();
    if(tid == 0) {
        *d.time += d.timesteps;
    }

    // Copy the water molecule back to global memory
    if(tid < d.num_waters) {
        d.flags[tid] = x;
        d.waters[tid] = w;
    }
}

void cpyLookupDevice(int **sourceTable, GPUData &d) {
    d.localLookup = new int*[hashDim * hashDim * hashDim];

    for(int i = 0; i < hashDim * hashDim * hashDim; i++) {
        d.localLookup[i] = (int *) cudaAllocate(maxNeighbors * sizeof(int));
        copyToDevice((void *) d.localLookup[i], (void *) sourceTable[i],
            maxNeighbors * sizeof(int));
    }
    d.lookupTable = (int**) cudaAllocate(hashDim * hashDim * hashDim * sizeof(int**));
    copyToDevice((void *) d.lookupTable, d.localLookup,
        hashDim * hashDim * hashDim * sizeof(int*));
}

void destroyLookupDevice(GPUData &d) {
    for(int i = 0; i < hashDim * hashDim * hashDim; i++) {
        hipFree(d.localLookup[i]);
    }
    hipFree(d.lookupTable);
    delete[] d.localLookup;
}

int main(void) {
    cout << "Starting GPU Simulation..." << endl;
    ofstream fout("test_output.csv");

    FCC lattice(D_cell, D_extra, P_expr);
    hipEvent_t start, stop;

    // Initialize PRNG seed for MNPs and waters
    std::random_device rd;
    XORShift<uint64_t> gen(time(NULL) + rd());

    // The simulation has 3 distinct components: the lattice, the water
    // molecules, and the nanoparticles
    vector<MNP_info> *mnps = lattice.init_mnps(gen);
    water_info *waters = lattice.init_molecules(bound, num_water, mnps, gen);
    Triple* linLattice = lattice.linearLattice();

    // Initialize the octree
    double max_product = 2e-6, max_g = 5, min_g = .002;
    uint64_t sTime = time(NULL);
    Octree tree(max_product, max_g, min_g, gen, mnps);
    uint64_t eTime = time(NULL) - sTime;
    std::cout << "Octree took " << eTime / 60 << ":";
    if (eTime % 60 < 10) std::cout << "0";
    std::cout << eTime % 60 << " to build." << std::endl << std::endl;

    GPUData d;
    setParameters(d);
    d.num_mnps = mnps->size();
    initOctree(&tree, d);

    cout << "Allocated GPU Octree!" << endl;
    int totalUniform =  num_uniform_doubles * num_water * sprintSteps;
    int totalNormal = num_normal_doubles * num_water * sprintSteps;
    int initTime = 0;

    // Allocations: Perform all allocations here
    HANDLE_ERROR(hipMalloc((void **) &(d.waters),
        num_water * sizeof(water_info)));
    HANDLE_ERROR(hipMalloc((void **) &(dev_lattice),
        num_cells * sizeof(Triple)));
    HANDLE_ERROR(hipMalloc((void **) &(d.flags),
        num_water * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **) &d.uniform_doubles,
        totalUniform*sizeof(double)));
    HANDLE_ERROR(hipMalloc((void **) &d.normal_doubles,
        totalNormal*sizeof(double)));
    HANDLE_ERROR(hipMalloc((void **) &d.time,
        sizeof(int)));

    d.lattice = (Triple*) cudaAllocate(sizeof(Triple) * num_cells);
    copyToDevice(d.lattice, linLattice, num_cells * sizeof(Triple));
    cpyLookupDevice(lattice.lookupTable, d);

    // Allocate the target array
    HANDLE_ERROR(hipMalloc((void **) &(d.magnetizations),
        num_blocks * (t / pfreq) * sizeof(double)));

    // Initialize performance timers
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    // Perform all memory copies here
    HANDLE_ERROR(hipMemcpy(d.waters, waters,
        sizeof(water_info) * num_water,
        hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d.time, &initTime,
        sizeof(int),
        hipMemcpyHostToDevice));

    int flags[num_water];
    double *magnetizations = new double[num_blocks * (t / pfreq)]; // Local magnetization target

    cout << "Kernel prepped!" << endl;

    // Run the kernel in sprints due to memory limits and timeout issues
    double time = 0;
    for(int i = 0; i < 2; i++) {
        cout << "Starting sprint " << (i+1) << "." << endl;
        getUniformDoubles(totalUniform, d.uniform_doubles);
        getNormalDoubles(totalNormal, d.normal_doubles);

        HANDLE_ERROR(hipEventRecord(start, 0));
        simulateWaters<<<num_blocks, threads_per_block>>>(d);

        HANDLE_ERROR(hipEventRecord(stop, 0));
        HANDLE_ERROR(hipEventSynchronize(stop));

        float elapsedTime;
        HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

        cout << "Kernel execution complete! Elapsed time: "
            << elapsedTime << " ms" << endl;

        // Copy back the array of flags to catch any errors
        HANDLE_ERROR(hipMemcpy(flags, d.flags,
            sizeof(int) * num_water,
            hipMemcpyDeviceToHost));

        bool success = true;
        for(int i = 0; i < num_water; i++) {
            if(flags[i] != t)
                success = false;
        }
        cout << "Success State: " << success << endl << "===========" << endl;

        // Copy back the array of magnetizations
        HANDLE_ERROR(hipMemcpy(magnetizations, d.magnetizations,
            num_blocks * (t / pfreq) * sizeof(double),
            hipMemcpyDeviceToHost));

        for(int j = 0; j < t / pfreq; j++) {
            double magSum = 0;
            for(int k = 0; k < num_blocks; k++) {
                magSum += magnetizations[j * num_blocks + k];
            }
            fout << time << "," << magSum << endl;

            time += 1e-3;
        }
    }

    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));
    destroyLookupDevice(d);
    finalizeGPU(d);

    delete[] linLattice;
    delete[] waters;
    delete[] magnetizations;
    delete mnps;
    fout.close();
}
