#include "hip/hip_runtime.h"
#define HIGH           5000000
#define MAX_MNPS       1000
#define M_PI           3.14159265358979323846

#include "cuda_helpers.h"

#include <stdio.h>
#include <time.h>
#include <dos.h>
#include <windows.h>
#include <stdlib.h>

#include <vector>
#include <fstream>
#include <thread>
#include <iostream>
#include "math.h"
#include "parameters.h"
#include "fcc_diffusion.h"
#include "gpu_random.h"
#include "octree.h"

using namespace std;
/**
 *  nvcc cuda_test.cu fcc_diffusion.cpp rand_walk.cpp octree.cpp -arch=sm_61 -lcurand -ccbin "
 * C:\Program Files (x86)\Microsoft Visual Studio 14.0\VC\bin\amd64_x86"
 */

#define threads_per_block 256
const int num_blocks = (num_water + threads_per_block - 1) / threads_per_block;

const double g = 42.5781e6;             // gyromagnetic ratio in MHz/T

// Each kernel execution handles AT MOST this many timesteps
const int sprintSteps = 10000;
const int num_uniform_doubles = 4; // # of uniform doubles per water per tstep
const int num_normal_doubles = 1;  // # of normal  doubles per water per tstep

__constant__ Triple dev_lattice[num_cells];

//==============================================================================

/**
 * Initialize a GPU verison of the octree from the CPU version.
 */
void initOctree(Octree *oct, GPUData &d) {
    // Initialize octree parameters

    vector<oct_node> *vec_nodes = oct->space;

    gpu_node *localTree = new gpu_node[vec_nodes->size()];
    MNP_info *localMNPs = new MNP_info[oct->mnps->size()];

    int mnp_idx = 0;
    for(int i = 0; i < vec_nodes->size(); i++) {
        oct_node current = (*vec_nodes)[i];
        localTree[i].mc = current.mc;

        for(int j = 0; j < 8; j++) {
            localTree[i].child[j] = current.child[j];
        }

        localTree[i].numResidents = current.resident->size();
        localTree[i].resIdx = mnp_idx;

        for(int j = 0; j < current.resident->size(); j++) {
            localMNPs[mnp_idx] = (*(current.resident))[j];
            mnp_idx++;
        }
    }

    HANDLE_ERROR(hipMalloc((void **) &(d.tree),
        vec_nodes->size() * sizeof(gpu_node)));
    HANDLE_ERROR(hipMalloc((void **) &(d.mnps),
        oct->mnps->size() * sizeof(MNP_info)));

    // Allocate arrays for morton codes
    HANDLE_ERROR(hipMalloc((void **) &(d.morton_x),
        256 * sizeof(uint32_t)));
    HANDLE_ERROR(hipMalloc((void **) &(d.morton_y),
        256 * sizeof(uint32_t)));
    HANDLE_ERROR(hipMalloc((void **) &(d.morton_z),
        256 * sizeof(uint32_t)));

    HANDLE_ERROR(hipMemcpy(d.tree, localTree,
        vec_nodes->size() * sizeof(gpu_node),
        hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d.mnps, localMNPs,
        oct->mnps->size() * sizeof(MNP_info),
        hipMemcpyHostToDevice));

    // Copy over the Morton code arrays
    HANDLE_ERROR(hipMemcpy(d.morton_x, morton_x,
        256 * sizeof(uint32_t),
        hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d.morton_y, morton_y,
        256 * sizeof(uint32_t),
        hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(d.morton_z, morton_z,
        256 * sizeof(uint32_t),
        hipMemcpyHostToDevice));

    delete[] localTree;
    delete[] localMNPs;
}

__device__ double dipole_field(double dx, double dy, double dz, double M)
{
    double d2x = dx*dx;
    double d2y = dy*dy;
    double d2z = dz*dz;
    double sum = d2x + d2y + d2z;
    double divisor = sum * sum * sqrt(sum);
    return M * 1e11 * (2*d2z - d2x - d2y) / divisor;
}

__device__ int morton_code(int depth, double x, double y, double z, GPUData &d) {

    double size = pow(2.0, depth);
    uint32_t idx_x = floor(x / d.bound * size);
    uint32_t idx_y = floor(y / d.bound * size);
    uint32_t idx_z = floor(z / d.bound * size);
    uint64_t answer = 0;
    // start by shifting the third byte, since we only look @ the first 21 bits
    if (depth > 16)
    {
        answer |=   d.morton_z[(idx_z >> 16) & 0xFF ] |
                    d.morton_y[(idx_y >> 16) & 0xFF ] |
                    d.morton_x[(idx_x >> 16) & 0xFF ];
        answer <<= 24;
    }

    // shift second byte
    if (depth > 8)
    {
        answer |=   d.morton_z[(idx_z >> 8) & 0xFF ] |
                    d.morton_y[(idx_y >> 8) & 0xFF ] |
                    d.morton_x[(idx_x >> 8) & 0xFF ];
        answer <<= 24;
    }

    // shift first byte
    answer |=   d.morton_z[(idx_z) & 0xFF ] |
                d.morton_y[(idx_y) & 0xFF ] |
                d.morton_x[(idx_x) & 0xFF ];
    return answer;
}

__device__ gpu_node* find_tree(double wx, double wy, double wz, GPUData &d)
{
    return d.tree + morton_code(d.min_depth, wx, wy, wz, d);
}

/*
 * Helper function to find the child index of a parent node at depth d that
 * holds the water molecule at (wx, wy, wz). This is done by taking the Morton
 * code of (wx, wy, wz) at depth d + 1 and returning the last 3 bits, which
 * would encode one additional level of depth.
 */
__device__ unsigned find_child(double wx, double wy, double wz, int d, GPUData &data)
{
    return morton_code(d + 1, wx, wy, wz, data) & 7;
}

// TODO: Check this function!
__device__ gpu_node* find_node(gpu_node *n, double wx, double wy, double wz, int d, GPUData &data) {
    while( ! (n->mc >> 63)) {
        // otherwise, navigate to the appropriate child and recurse
        unsigned child_no = find_child(wx, wy, wz, d, data);
        d += 1;
        n += n->child[child_no].idx;
    }
    return n;
}

__device__ gpu_node* get_voxel(water_info *w, GPUData &d) {
    double wx = w->x, wy = w->y, wz = w->z;
    return find_node(find_tree(wx, wy, wz, d), wx, wy, wz, d.min_depth, d);
}

/**
 * Returns the B field at the location of a particular water molecule
 */
__device__ double get_field(water_info *w, GPUData &d) {
    double wx = w->x, wy = w->y, wz = w->z;
    gpu_node *leaf = get_voxel(w, d);

    uint64_t depth = 0, mc = (leaf->mc << 1) >> 1;
    while (mc >>= 3) depth++;

    // use Morton code's depth to find child index to find value of B to return
    unsigned child_no = find_child(wx, wy, wz, depth, d);
    double B = (double)leaf->child[child_no].B;

    // add in contributions from resident MNPs zeroed out during construction
    for(int i = leaf->resIdx; i < leaf->resIdx + leaf->numResidents; i++) {
        MNP_info *np = d.mnps + i;
        B += dipole_field(wx - np->x, wy - np->y, wz - np->z, np->M);
    }

    return B;
}

//==============================================================================

void finalizeGPU(GPUData &d) {
    hipFree(d.waters);
    hipFree(d.flags);
    hipFree(d.uniform_doubles);
    hipFree(d.normal_doubles);
    hipFree(d.magnetizations);
    hipFree(d.mnps);
    hipFree(d.tree);

    hipFree(d.morton_x);
    hipFree(d.morton_y);
    hipFree(d.morton_z);
}

void setParameters(GPUData &d) {
    // Initialize constants for the GPU
    d.in_stdev = sqrt(pi * D_cell * tau);
    d.out_stdev = sqrt(pi * D_extra * tau);

    d.reflectIO = 1 - sqrt(tau / (6*D_cell)) * 4 * P_expr;
    d.reflectOI = 1 - ((1 - d.reflectIO) * sqrt(D_cell/D_extra));

    d.num_cells = num_cells;
    d.num_waters = num_water;
    d.timesteps = sprintSteps;
    d.cell_r = cell_r;
    d.bound = bound;
    d.nBlocks = num_blocks;
    d.g = g;
    d.tau = tau;
    d.bound = bound;
}

/**
 * Trivial implementation of nearest cell finder
 */
__device__ void updateNearest(water_info *w, GPUData &d) {
    int cIndex = 0;
    double cDist = HIGH;
    w->in_cell = false;
    for(int i = 0; i < 172; i++) {
        double dx = w->x - dev_lattice[i].x;
        double dy = w->y - dev_lattice[i].y;
        double dz = w->z - dev_lattice[i].z;
        if(NORMSQ(dx, dy, dz) < cDist) {
            cDist = NORMSQ(dx, dy, dz);
            cIndex = i;
        }
    }
    if(cDist < d.cell_r * d.cell_r)
        w->in_cell = true;
    w->nearest = cIndex;
}

__device__ bool cell_reflect(water_info *i, water_info *f, int tStep, GPUData &d) {
    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    double coin = d.uniform_doubles[tStep * d.num_waters * 4 + tid * 4 + 3];
    bool flip = (i->in_cell && (! f->in_cell) && coin < d.reflectIO)
                    || ((! i->in_cell) && f->in_cell && coin < d.reflectOI);
    return flip;
}

__device__ bool mnp_reflect(water_info *w, int num_mnps, MNP_info *mnps) {
    return false; // TODO: Fix this!!!
}

__device__ water_info rand_displacement(int tid, int tStep, water_info *w, GPUData &d) {
    water_info disp;
    double norm = abs(d.normal_doubles[tStep * d.num_waters + tid]);
    int baseU = tStep * d.num_waters * 4 + tid * 4;

    disp.x = d.uniform_doubles[baseU];
    disp.y = d.uniform_doubles[baseU + 1];
    disp.z = d.uniform_doubles[baseU + 2];

    if(w->in_cell) {
        norm *= d.in_stdev;
    }
    else {
        norm *= d.out_stdev;
    }

    double nConstant = norm / NORMSQ(disp.x, disp.y, disp.z);

    disp.x *= nConstant;
    disp.y *= nConstant;
    disp.z *= nConstant;

    return disp;
}

__device__ void boundary_conditions(water_info *w, GPUData &d) {
    w->x = fmod(w->x + d.bound, d.bound);
    w->y = fmod(w->y + d.bound, d.bound);
    w->z = fmod(w->z + d.bound, d.bound);
}

__device__ void accumulatePhase(water_info *w, GPUData &d) {
    double B = get_field(w, d);
    w->phase += B * 2 * M_PI * d.g * d.tau * 1e-3;
}
// END PHASE ACCUMULATION FUNCTIONS

__device__ void carrPurcellFlip(water_info *w, int tStep) {

}

__device__ void sumMagnetizations(double *input, int tStep, double *target) {

}

__global__ void simulateWaters(GPUData d)  {
    __shared__ double mags[threads_per_block];

    int tid = threadIdx.x + blockIdx.x * blockDim.x;
    water_info w;
    int x = 0;

    if(tid < d.num_waters) {
        // Copy water to chip memory
        w = d.waters[tid];
        d.flags[tid] = -1;
        updateNearest(&w, d);
    }

    for(int i = 0; i < d.timesteps; i++) {
        if(tid < d.num_waters) {
            x++;
            water_info init = w;
            water_info disp = rand_displacement(tid, i, &w, d);
            w.x += disp.x;
            w.y += disp.y;
            w.z += disp.z;
            boundary_conditions(&w, d);
            //updateNearest(&w, d);

            // Check cell boundary / MNP reflection
            if(cell_reflect(&init, &w, i, d))
                w = init;

            accumulatePhase(&w, d);
            carrPurcellFlip(&w, i);

            /*
            // Copy the magnetization to shared memory
            mags[i] = cos(w.phase);

            // Perform a memory reduction
            sumMagnetizations(mags, i, d.magnetizations);
            */
        }
    }

    if(tid < d.num_waters) {
        d.flags[tid] = x;
        d.waters[tid] = w;
    }
}

int main(void) {
    cout << "Starting GPU Simulation..." << endl;
    FCC lattice(D_cell, D_extra, P_expr);
    hipEvent_t start, stop;

    // Initialize PRNG seed for MNPs and waters
    std::random_device rd;
    XORShift<uint64_t> gen(time(NULL) + rd());

    // The simulation has 3 distinct components: the lattice, the water
    // molecules, and the nanoparticles
    vector<MNP_info> *mnps = lattice.init_mnps(gen);
    water_info *waters = lattice.init_molecules(bound, num_water, mnps, gen);
    Triple* linLattice = lattice.linearLattice();
    int* lookupTable = lattice.linearLookupTable();

    // Initialize the octree
    double max_product = 2e-6, max_g = 5, min_g = .002;
    uint64_t sTime = time(NULL);
    Octree tree(max_product, max_g, min_g, gen, mnps);
    uint64_t eTime = time(NULL) - sTime;
    std::cout << "Octree took " << eTime / 60 << ":";
    if (eTime % 60 < 10) std::cout << "0";
    std::cout << eTime % 60 << " to build." << std::endl << std::endl;

    GPUData d;
    setParameters(d);
    d.num_mnps = mnps->size();
    //initOctree(&tree, d);

    cout << "Allocated GPU Octree!" << endl;
    int totalUniform =  num_uniform_doubles * num_water * sprintSteps;
    int totalNormal = num_normal_doubles * num_water * sprintSteps;
    // Allocations: Perform all allocations here
    HANDLE_ERROR(hipMalloc((void **) &(d.waters),
        num_water * sizeof(water_info)));
    HANDLE_ERROR(hipMalloc((void **) &(dev_lattice),
        num_cells * sizeof(Triple)));
    HANDLE_ERROR(hipMalloc((void **) &(d.flags),
        num_water * sizeof(int)));
    HANDLE_ERROR(hipMalloc((void **) &d.uniform_doubles,
        totalUniform*sizeof(double)));
    HANDLE_ERROR(hipMalloc((void **) &d.normal_doubles,
        totalNormal*sizeof(double)));
    HANDLE_ERROR(hipMalloc((void **) &(d.magnetizations),
        num_blocks * t * sizeof(double)));

    // Initialize performance timers
    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    // Perform all memory copies here
    HANDLE_ERROR(hipMemcpy(d.waters, waters,
        sizeof(water_info) * num_water,
        hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(dev_lattice), linLattice,
        sizeof(Triple) * num_cells));

    int flags[num_water];

    cout << "Kernel prepped!" << endl;

    // Run the kernel in sprints due to memory limits and timeout issues
    for(int i = 0; i < 10; i++) {
        cout << "Starting sprint " << (i+1) << "." << endl;
        getUniformDoubles(totalUniform, d.uniform_doubles);
        getNormalDoubles(totalNormal, d.normal_doubles);

        HANDLE_ERROR(hipEventRecord(start, 0));
        simulateWaters<<<num_blocks, threads_per_block>>>(d);

        HANDLE_ERROR(hipEventRecord(stop, 0));
        HANDLE_ERROR(hipEventSynchronize(stop));

        float elapsedTime;
        HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

        cout << "Kernel execution complete! Elapsed time: "
            << elapsedTime << " ms" << endl;

        // Copy back the array of flags to catch any errors
        HANDLE_ERROR(hipMemcpy(flags, d.flags,
            sizeof(int) * num_water,
            hipMemcpyDeviceToHost));

        bool success = true;
        for(int i = 0; i < num_water; i++) {
            if(flags[i] != t)
                success = false;
        }
        cout << "Success State: " << success << endl << "===========" << endl;
    }

    finalizeGPU(d);
    HANDLE_ERROR(hipEventDestroy(start));
    HANDLE_ERROR(hipEventDestroy(stop));

    delete[] linLattice;
    delete[] lookupTable;
    delete[] waters;
    delete mnps;
}
