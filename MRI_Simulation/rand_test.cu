
/*
 * This program uses the host CURAND API to generate 100
 * pseudorandom floats.
 */
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "cuda_helpers.h"

using namespace std;

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

int main(int argc, char *argv[])
{
    size_t n = 100000000;
    size_t i;
    hiprandGenerator_t gen;
    double *devData, *hostData;

    /* Allocate n floats on host */
    hostData = (double *) calloc(n, sizeof(double));
    hipEvent_t start, stop;

    /* Allocate n floats on device */
    CUDA_CALL(hipMalloc((void **)&devData, n*sizeof(double)));

    /* Create pseudo-random number generator */
    /*CURAND_CALL(hiprandCreateGenerator(&gen,
                HIPRAND_RNG_PSEUDO_DEFAULT));*/
    CURAND_CALL(hiprandCreateGenerator(&gen,
                HIPRAND_RNG_PSEUDO_XORWOW));

    /* Set seed */
    CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen,
                time(NULL)));

    HANDLE_ERROR(hipEventCreate(&start));
    HANDLE_ERROR(hipEventCreate(&stop));

    /* invoke the kernel to get some random numbers */
    HANDLE_ERROR(hipEventRecord(start, 0));

    cout << "Generating random numbers!" << endl;
    /* Generate n floats on device */
    CURAND_CALL(hiprandGenerateUniformDouble(gen, devData, n));
    HANDLE_ERROR(hipEventRecord(stop, 0));
    HANDLE_ERROR(hipEventSynchronize(stop));

    float elapsedTime;
    HANDLE_ERROR(hipEventElapsedTime(&elapsedTime, start, stop));

    cout << "RNG complete! Elapsed time: "
      << elapsedTime << " ms" << endl;

    /* Copy device memory to host */
    CUDA_CALL(hipMemcpy(hostData, devData, n * sizeof(double),
        hipMemcpyDeviceToHost));

    /* Show result */
    for(i = 0; i < 200; i++) {
        printf("%1.8f ", hostData[i]);
    }
    printf("\n");

    /* Cleanup */
    CURAND_CALL(hiprandDestroyGenerator(gen));
    CUDA_CALL(hipFree(devData));
    return EXIT_SUCCESS;
}
